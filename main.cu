#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sys/stat.h>
#include "components.h"
#include "Board.h"
#include "Player.h"
#include "State.h"
#include "util.h"
#include <chrono>
#include <cfloat>
#include <ctime>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

long runProgram(float performanceMeasure) {
	ifstream inputFile;
	fstream playDataFile;
	inputFile.open("./input.txt");
	StateVector initState{};
	string executionType = "SINGLE", s = "";
	char team = 'B';
	bool randNeeded;
	float timeLeft = 100.0, previousScore = -1.0;
	int counter = 0;
	while (inputFile >> s) {
		switch (counter) {
		case 0:
			// First line: Type of execution. By default is SINGLE
			if (s.length() > 0) {
				executionType = s;
			}
			break;
		case 1:
			// Second line: Team
			team = s == "WHITE" ? 'W' : 'B';
			break;
		case 2:
			// Third line: Number of seconds remaining
			try {
				timeLeft = stof(s);
			}
			catch (...) {
				// If there is an error in the input then timeLeft is default
			}
			break;
		default:
			// The board
			vector<char> row(s.begin(), s.end());
			initState.push_back(row);
		}
		counter++;
	}
	Board board = Board(initState);
	/*
		If the execution type is a game, then check for a playdata.txt
		The playdata.txt contains the time you can take for a move, and maybe a sequence of steps.
	*/
	State* currState = new State(initState, { {} }, NULL, true);
	currState->computeScore(team, board.getBase(team));

	if (executionType == "GAME") {
		struct stat playFile;
		if (stat("./playdata.txt", &playFile) != -1) {
			// File exists, read it.
			playDataFile.open("./playdata.txt");
			playDataFile >> s;
			float timeLeftFromPlayData = stof(s);
			playDataFile >> s;
			previousScore = stof(s);
			if (timeLeftFromPlayData > timeLeft) {
				// We don't have the time we think we have
				// Try to squeeze out 10 moves from the remaining time
				timeLeft = timeLeft / 10;
			}
			else {
				timeLeft = timeLeftFromPlayData;
			}
		}
		else {
			timeLeft = timeLeft / 110;
		}
		// Average game length of 110 moves
	}
	int depth = getDepth(timeLeft, performanceMeasure, currState->getScore());
	PositionsVector playerPositions = getPositions(initState, team);
	Player player = Player(team, playerPositions);
	/*
		Generate the minmax tree with the following attributes:
			The current State
			How deep can the player jump
			The number of turns
			The locations of the player's points
			Alpha and Beta [For Alpha-Beta Pruning]
	*/
	auto start = chrono::high_resolution_clock::now();
	currState = board.generateMinMaxTree(currState, 2, player.getLocations(), -FLT_MAX + 1, FLT_MAX, true);

	// Get the argmax of all alphabetas of currState's children
	vector<State*> children = currState->getChildren();
	float maxChildScore = -FLT_MAX;
	int maxChildLoc = -1;

	for (int i = 0; i < children.size(); i++) {
		if (children[i]->getAlphaBetaPrediction() > maxChildScore) {
			maxChildScore = children[i]->getAlphaBetaPrediction();
			maxChildLoc = i;
		}
	}
	string result;
	if (maxChildLoc == -1) {
		// No valid moves, gracefully exit
		result = "NO VALID MOVES";
	}
	else {
		State* desiredChild = children[maxChildLoc];
		if (abs(previousScore - desiredChild->getAlphaBetaPrediction()) < 1.5) {
			srand(time(NULL));
			int someNumber = rand();
			maxChildLoc = someNumber % children.size();
			desiredChild = children[maxChildLoc];
		}
		result = generateString(desiredChild->getPositions(), desiredChild->isStateAJump());
	}
	ofstream outFile;
	outFile.open("./output.txt");
	outFile << result;
	outFile.close();
	if (executionType == "GAME") {
		// Some persistent data we may be able to use
		// Store the actual playtime.
		playDataFile.open("./playdata.txt", fstream::out);
		char* timeLeftString = (char*)malloc(20 * sizeof(char));
		snprintf(timeLeftString, 20, "%.4f", timeLeft);
		playDataFile << timeLeftString << endl;
		playDataFile << currState->getScore();
		playDataFile.close();
	}
	auto end = chrono::high_resolution_clock::now();
	long actual = chrono::duration_cast<chrono::microseconds>(end - start).count();
	return actual;
}

int execProg() {
	float performanceMeasure = calibrate();
	runProgram(performanceMeasure);
	return 0;
}