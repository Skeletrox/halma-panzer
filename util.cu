#include "hip/hip_runtime.h"
/*
	Utility file. Define utility functions such as position initialization here.
*/
#include "util.h"
#include <math.h>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

float max(float a, float b) {
	return a > b ? a : b;
}

float min(float a, float b) {
	return a < b ? a : b;
}


// Returns the positions of all the pieces of a certain team.
PositionsVector getPositions(StateVector boardState, char team) {
	PositionsVector neededVector{};
	// Iterate through the 16x16 board
	int count = 0;
	bool breakable = false;
	for (int i = 0; i < boardState.size(); i++) {
		for (int j = 0; j < boardState[i].size(); j++) {
			if (boardState[i][j] == team) {
				neededVector.push_back({ j, i });
				count++;
				if (count == 19) {
					breakable = true;
					break;
				}
			}
		}
		if (breakable) {
			break;
		}
	}
	return neededVector;
}

// Returns true if the move involves a jump, useful for checking if recursion needs to be handled.
bool isJump(PositionsVector positions) {
	return (abs(positions[0][0] - positions[1][0]) > 1 || abs(positions[0][1] - positions[1][1]) > 1);
}

// Utility function, defined as the distance from (x, y) to y = x
float utility(int x, int y) {
	float numerator = abs(float(x) - float(y));
	return ( numerator / float(sqrt(2)));
}

void doSomething(int x) {
	if (x == 0) {
		return;
	}
	doSomething(x / 2);
}

long calibrate() {
	auto start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < 1000; i++) {
		doSomething(i);
	}
	auto end = std::chrono::high_resolution_clock::now();
	long diff = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	// Avoid zeros for powerful machines
	return diff + 1;
}

int getDepth(float timeRemaining, long calibratedValue, float currentScore) {
	/*
		The hard time limits are as follows:
			The following time limits are generated on a calibration factor of 1,
			i.e. incrementing a variable 1000 times takes 1 miecrosecond

			The time remaining can be expressed in microseconds, and multiply time taken calibratedValue, as a slower computer
			can cause our values to be awry.
	*/
	float timeRemainingMicrosec = timeRemaining * 1000000;
	std::cout << "Remaining time: " << timeRemainingMicrosec << " and calibrated value " << calibratedValue <<  " and score " << currentScore << std::endl;
	std::vector<long> timesTerminal{ 7416, 24016, 129892, 1457973, 5266859, 51633889 }, timesMidway = { 1419, 77200, 359061, 6874063, 46678036 }, times;
	times = currentScore < 127 ? timesTerminal : timesMidway;
	for (int i = 0; i < times.size()-1; i++) {
		if (timeRemainingMicrosec < (times[i + 1] * calibratedValue)) {
			// We don't have enough time to deepen to the next level; stop here
			std::cout << "Expected duration: " << times[i] << " " << i << std::endl;
			return i+1; // Zero-indexed array
		}
	}
	std::cout << "Expected duration: " << times[times.size() - 1] << " " << times.size() << std::endl;
	return times.size();
}

bool isIllegal(int xStart, int yStart, int xEnd, int yEnd, PositionsVector baseAnchors, char team) {
	if (team == 'B') {
		// 1. Your own base is the baseAnchors. Make sure that your piece, if not in your base, does not jump back
		// 2. Your opponent's base is at 15 - baseAnchors. Do not jump out of it.
		if (!found(xStart, yStart, baseAnchors, false) && found(xEnd, yEnd, baseAnchors, false)) { // Case 1
			// std::cout << "Black base cowardice" << std::endl;
			return true;
		} else if (found(xStart, yStart, baseAnchors, true) && !found(xEnd, yEnd, baseAnchors, true)) { // Case 2
			// std::cout << "Black base escapism" << std::endl;
			return true;
		} else if (found(xStart, yStart, baseAnchors, false) && found(xEnd, yEnd, baseAnchors, false) && (isOrderedAs(xStart, xEnd, baseAnchors[0][0], false) || isOrderedAs(yStart, yEnd, baseAnchors[0][1], false))) {
			// std::cout << "Black base cowardice within" << std::endl;
			return true;
		}
	}
	else {
		// The above lines switched in context.
		if (!found(xStart, yStart, baseAnchors, true) && found(xEnd, yEnd, baseAnchors, true)) { // Case 1
			// std::cout << "White base cowardice" << std::endl;
			return true;
		} else if (found(xStart, yStart, baseAnchors, false) && !found(xEnd, yEnd, baseAnchors, false)) { // Case 2
			// std::cout << "White base escapism" << std::endl;
			return true;
		} else if (found(xStart, yStart, baseAnchors, true) && found(xEnd, yEnd, baseAnchors, true) && (isOrderedAs(xStart, xEnd, baseAnchors[0][0], true) || isOrderedAs(yStart, yEnd, baseAnchors[0][1], true))) {
			// std::cout << "White base cowardice within" << std::endl;
			return true;
		}
	}
	// Legal move only
	return false;
}

bool found(int x, int y, PositionsVector baseAnchors, bool reverse) {
	for (std::array<int, 2> b : baseAnchors) {
		if ((x == (reverse ? 15 - b[0] : b[0])) && (y == (reverse ? 15 - b[1] : b[1]))) {
			return true;
		}
	}
	return false;
}

/*
	Returns true if second is either at third or on the way between first and third.
*/
bool isOrderedAs(int first, int second, int third, bool reverse) {
	if (reverse) {
		third = 15 - third;
		// std::cout << "First: " << first << " second: " << second << " third: " << third << std::endl;
		return first < second && second <= third;
	}
	// std::cout << "First: " << first << " second: " << second << " third: " << third << std::endl;
	return first > second && second >= third;
}

// Returns the diagonal mirror of the positions along y + x = 15
PositionsVector getMirror(PositionsVector original) {
	PositionsVector mirror;
	for (std::array<int, 2> o : original) {
		std::array<int, 2> v{ 15 - o[0], 15 - o[1] };
		mirror.push_back(v);
	}
	return mirror;
}

PositionsSet getMirrorSet(PositionsVector original) {
	PositionsSet mirror;
	for (std::array<int, 2> o : original) {
		std::array<int, 2> v{ 15 - o[0], 15 - o[1] };
		mirror.insert(v);
	}
	return mirror;
}

void printPositions(PositionsVector positions) {
	for (std::array<int, 2> p : positions) {
		std::cout << p[0] << "," << p[1] << " ";
	}
	std::cout << std::endl;
}

void printState(StateVector state) {
	for (int i = 0; i < state.size(); i++) {
		for (int j = 0; j < state[i].size(); j++) {
			std::cout << state[i][j];
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

std::string generateString(PositionsVector positions, bool isJump) {
	std::string returnable = "";
	for (int i = 0; i < positions.size() - 1; i++) {
		if (isJump) {
			returnable += "J ";
		}
		else {
			returnable += "E ";
		}
		std::array<int, 2> first = positions[i], second = positions[i + 1];
		char* currentLine = (char*)malloc(40 * sizeof(char));
		snprintf(currentLine, 40, "%d,%d %d,%d\n", first[0], first[1], second[0], second[1]);
		returnable.append(currentLine);
	}
	return returnable;
}