#include "Player.h"
#include "State.h"
#include <array>
#include "hip/hip_runtime.h"
#include ""

Player::Player(char argTeam, PositionsVector argLocations) {
	team = argTeam;
	score = 0;
	locations = argLocations;
}

char Player::getTeam() {
	return team;
}

float Player::getScore() {
	return score;
}

PositionsVector Player::makeMove(StateVector state) {
	PositionsVector p;
	// Given a state, choose the appropriate movement and get a score
	// Get all future states for the current state
	return p;
}

PositionsVector Player::getLocations() {
	return locations;
}


